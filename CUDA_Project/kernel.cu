#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <stdio.h>
#include <cstdlib>
#include ""



#define BLOCK_SIZE 256

__global__ void matchPattern(const char* text, int textLen, const char* pattern, int patternLen, int* positions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx + patternLen <= textLen) {
        bool match = true;
        for (int i = 0; i < patternLen && match; ++i) {
            if (text[idx + i] != pattern[i]) {
                match = false;
            }
        }
        if (match) {
            positions[idx] = 1;
        }
    }
}

void printMatches(const int* positions, int textLen, const std::string& text, const std::string& pattern) {
    for (int i = 0; i < textLen; ++i) {
        if (positions[i] == 1) {
            std::cout << "Pattern \"" << pattern << "\" found at position " << i << " in text \"" << text << "\"" << std::endl;
        }
    }
}

int main() {
    std::string text = "hello world, CUDA welcome to CUDA programming";
    std::string pattern = "CUDA";
    int textLen = text.size();
    int patternLen = pattern.size();

    char* d_text, * d_pattern;
    int* d_positions, * positions;

    hipMalloc(&d_text, textLen);
    hipMalloc(&d_pattern, patternLen);
    hipMalloc(&d_positions, textLen * sizeof(int));

    hipMemcpy(d_text, text.c_str(), textLen, hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern.c_str(), patternLen, hipMemcpyHostToDevice);

    // Initialize positions array to 0
    positions = new int[textLen]();
    hipMemcpy(d_positions, positions, textLen * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (textLen + BLOCK_SIZE - 1) / BLOCK_SIZE;
    matchPattern << <blocks, BLOCK_SIZE >> > (d_text, textLen, d_pattern, patternLen, d_positions);

    hipMemcpy(positions, d_positions, textLen * sizeof(int), hipMemcpyDeviceToHost);

    printMatches(positions, textLen, text, pattern);

    hipFree(d_text);
    hipFree(d_pattern);
    hipFree(d_positions);
    delete[] positions;

    return 0;
}
